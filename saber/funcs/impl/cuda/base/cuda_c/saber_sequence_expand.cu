#include "hip/hip_runtime.h"
#include "saber/funcs/impl/cuda/saber_sequence_expand.h"
#include "hip/hip_fp16.h"

namespace anakin{
namespace saber{

template<typename Dtype>
__global__ void ker_relu_fwd(Dtype * out_data,
                   const Dtype* in_data, const int count, Dtype neg_slop,
                   int in_n, int in_c, int in_h, int in_w,
                   int in_n_stride, int in_c_stride, int in_h_stride, int in_w_stride,
                   int out_n_stride, int out_c_stride, int out_h_stride, int out_w_stride) {
    CUDA_KERNEL_LOOP(tid, count){
        int w =  tid % in_w;
        int h = (tid / (in_w)) % in_h;
        int c = (tid / (in_h * in_w)) % in_c;
        int n = (tid / (in_c * in_h * in_w)) % in_n;

        int in_idx = n * in_n_stride
                   + c * in_c_stride
                   + h * in_h_stride
                   + w * in_w_stride;

        int out_idx =  n * out_n_stride
                     + c * out_c_stride
                     + h * out_h_stride
                     + w * out_w_stride;

        Dtype in_var = in_data[in_idx];
        out_data[out_idx] = in_var > Dtype(0) ? in_var : in_var * neg_slop;
    }
}


template <>
SaberStatus SaberSequenceExpand<NV, AK_FLOAT, AK_FLOAT, AK_FLOAT, \
        NCHW, NCHW, NCHW>::dispatch( \
        const std::vector<DataTensor_in*>& inputs,
        std::vector<DataTensor_out*>& outputs,
        SequenceExpandParam<OpTensor>& param) {

    Shape in_shape = inputs[0]->valid_shape();
    Shape out_shape = outputs[0]->valid_shape();

    const InDataType *in_data = (const InDataType*)inputs[0]->data();
    OutDataType *out_data = (OutDataType*)outputs[0]->mutable_data();

    const int count = inputs[0]->valid_size();
    hipStream_t hip_stream = this->_ctx->get_compute_stream();

    InDataType negative_slope = param.negative_slope;
    InDataType coef = param.coef;

    ker_relu_fwd<InDataType>
            <<<CUDA_GET_BLOCKS(count), CUDA_NUM_THREADS, 0, hip_stream>>>(
            out_data, in_data, count, negative_slope,
            in_shape[0], in_shape[1], in_shape[2], in_shape[3],
            stride_in[0], stride_in[1], stride_in[2], stride_in[3],
            stride_out[0], stride_out[1], stride_out[2], stride_out[3]);

    CUDA_POST_KERNEL_CHECK;
    return SaberSuccess;
}

}
}
