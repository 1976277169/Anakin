#include "hip/hip_runtime.h"
#include "saber/core/tensor_op.h"
#include <limits>

namespace anakin{

namespace saber{

template <typename Dtype>
__global__ void set_device_data(Dtype* data_ptr, Dtype value, int size){
    CUDA_KERNEL_LOOP(index, size){
        data_ptr[index] = value;
    }
}

template <typename Dtype>
__global__ void print_device_data(const Dtype* data_ptr, int size, int width){
    for (int i = 0; i < size; i++){
        printf("%.2f ", static_cast<float>(data_ptr[i]));
        if ((i + 1) % width == 0){
            printf("\n");
        }
    }
    printf("\n");
}

template <typename Dtype>
__global__ void cuda_cvt_data(const float* src, Dtype* dst, Dtype scale, int size){
    CUDA_KERNEL_LOOP(index, size){
        dst[index] = static_cast<Dtype>(src[index] * scale);
    }
}

template <class Tensor_t>
void fill_tensor_device_const(Tensor_t& tensor, \
    typename Tensor_t::FDtype value, \
    typename Tensor_t::API::stream_t stream){

    typedef typename Tensor_t::FDtype Dtype;
    Dtype* data_ptr = tensor.mutable_data();
    int size = tensor.size();
    set_device_data<<<CUDA_GET_BLOCKS(size), CUDA_NUM_THREADS, 0, stream>>>(data_ptr, value, size);
    CUDA_POST_KERNEL_CHECK;
};


template <class Tensor_t>
void fill_tensor_device_rand(Tensor_t& tensor, typename Tensor_t::API::stream_t stream) {

    typedef typename Tensor_t::FDtype Dtype;
    Dtype* data_ptr = tensor.mutable_data();
    int size = tensor.size();

    float* data_f;
    hipMalloc(&data_f, size * sizeof(float));

    hiprandGenerator_t gen;
    CHECK_EQ(hiprandCreateGenerator(&gen, HIPRAND_RNG_PSEUDO_DEFAULT), HIPRAND_STATUS_SUCCESS);
    CHECK_EQ(hiprandSetPseudoRandomGeneratorSeed(gen, rand()), HIPRAND_STATUS_SUCCESS);
    CHECK_EQ(hiprandGenerateUniform(gen, data_f, size), HIPRAND_STATUS_SUCCESS);
    CHECK_EQ(hiprandDestroyGenerator(gen), HIPRAND_STATUS_SUCCESS);

    Dtype scale = std::numeric_limits<Dtype>::max();

    cuda_cvt_data<<<CUDA_GET_BLOCKS(size), CUDA_NUM_THREADS, 0, stream>>>(data_f, data_ptr, scale, size);
    hipDeviceSynchronize();
    hipFree(data_f);

    CUDA_POST_KERNEL_CHECK;
};

template <class Tensor_t>
void fill_tensor_device_rand(Tensor_t& tensor, typename Tensor_t::FDtype vstart, \
    typename Tensor_t::FDtype vend, typename Tensor_t::API::stream_t stream) {

    typedef typename Tensor_t::FDtype Dtype;
    Dtype* data_ptr = tensor.mutable_data();
    int size = tensor.size();

    float* data_f;
    hipMalloc(&data_f, size * sizeof(float));

    hiprandGenerator_t gen;
    CHECK_EQ(hiprandCreateGenerator(&gen, HIPRAND_RNG_PSEUDO_DEFAULT), HIPRAND_STATUS_SUCCESS);
    CHECK_EQ(hiprandSetPseudoRandomGeneratorSeed(gen, rand()), HIPRAND_STATUS_SUCCESS);
    CHECK_EQ(hiprandGenerateUniform(gen, data_f, size), HIPRAND_STATUS_SUCCESS);
    CHECK_EQ(hiprandDestroyGenerator(gen), HIPRAND_STATUS_SUCCESS);

    Dtype scale = vend - vstart;

    cuda_cvt_data<<<CUDA_GET_BLOCKS(size), CUDA_NUM_THREADS, 0, stream>>>(data_f, data_ptr, scale, size);
    hipDeviceSynchronize();
    hipFree(data_f);

    CUDA_POST_KERNEL_CHECK;
};

template <class Tensor_t>
void print_tensor_device(Tensor_t& tensor, typename Tensor_t::API::stream_t stream){

    typedef typename Tensor_t::FDtype Dtype;
    LOG(INFO) << "device tensor size: " << tensor.size();
    const Dtype* data_ptr = tensor.data();
    int size = tensor.size();
    print_device_data<<<1, 1, 0, stream>>>(data_ptr, size, tensor.width());
    hipDeviceSynchronize();
    CUDA_POST_KERNEL_CHECK;
};

template void fill_tensor_device_const<Tensor<NV>>(Tensor<NV>& tensor, Tensor<NV>::FDtype value, \
        typename TargetWrapper<NV>::stream_t stream);
template void fill_tensor_device_rand<Tensor<NV>>(Tensor<NV>& tensor, typename TargetWrapper<NV>::stream_t stream);
template void fill_tensor_device_rand<Tensor<NV>>(Tensor<NV>& tensor, Tensor<NV>::FDtype vstart, \
        Tensor<NV>::FDtype vend, typename TargetWrapper<NV>::stream_t stream);
template void print_tensor_device<Tensor<NV>>(Tensor<NV>& tensor, typename TargetWrapper<NV>::stream_t stream);

} //namespace saber

} //namespace anakin
